#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include "kernels.h"

extern void forward_1d_fft(fftPrecision** data, int Ng, int nlocal){

    hipfftHandle plan;

    int nFFTs = nlocal / Ng;

    #ifdef doublePrecision

    if (hipfftPlan1d(&plan, Ng, HIPFFT_Z2Z, nFFTs) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;	
    }

    if (hipfftExecZ2Z(plan, (cudafftPrecision*)data[0], (cudafftPrecision*)data[0], HIPFFT_FORWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecZ2Z Forward failed\n");
        return;	
    }

    #else

    if (hipfftPlan1d(&plan, Ng, HIPFFT_C2C, nFFTs) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;	
    }

    if (hipfftExecC2C(plan, (cudafftPrecision*)data[0], (cudafftPrecision*)data[0], HIPFFT_FORWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecC2C Forward failed\n");
        return;	
    }
    #endif

    hipDeviceSynchronize();

}

extern void inverse_1d_fft(fftPrecision** data, int Ng, int nlocal){

    hipfftHandle plan;

    int nFFTs = nlocal / Ng;

    #ifdef doublePrecision

    if (hipfftPlan1d(&plan, Ng, HIPFFT_Z2Z, nFFTs) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;	
    }

    if (hipfftExecZ2Z(plan, (cudafftPrecision*)data[0], (cudafftPrecision*)data[0], HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecZ2Z Forward failed\n");
        return;	
    }

    #else

    if (hipfftPlan1d(&plan, Ng, HIPFFT_C2C, nFFTs) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;	
    }

    if (hipfftExecC2C(plan, (cudafftPrecision*)data[0], (cudafftPrecision*)data[0], HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecC2C Forward failed\n");
        return;	
    }

    #endif

    hipDeviceSynchronize();

    }

__global__
void scale_fft(fftPrecision* __restrict out, const fftPrecision* __restrict data, int Ng, int nlocal){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    fftPrecision scale = (fftPrecision)(Ng * Ng * Ng);

    if (idx < nlocal){

        out[idx*2] = data[idx*2] / scale;
        out[idx*2 + 1] = data[idx*2 + 1] / scale;

    }

}

__global__
void fast_copy_fft(fftPrecision* __restrict out, const fftPrecision* __restrict data, int nlocal){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < nlocal){

        out[idx*2] = data[idx*2];
        out[idx*2 + 1] = data[idx*2 + 1];

    }

}

extern void launch_scale_fft(fftPrecision** out, fftPrecision** data, int Ng, int nlocal, int blockSize){

    int numBlocks = (nlocal + blockSize - 1) / blockSize;

    scale_fft<<<numBlocks,blockSize>>>(out[0],data[0],Ng,nlocal);

    hipDeviceSynchronize();

}

extern void launch_fast_copy_fft(fftPrecision** out, fftPrecision** data, int nlocal, int blockSize){

    int numBlocks = (nlocal + blockSize - 1) / blockSize;

    fast_copy_fft<<<numBlocks,blockSize>>>(out[0],data[0],nlocal);

    hipDeviceSynchronize();

}