#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include "kernels.h"

extern void forward_1d_fft(fftPrecision** data, int Ng, int nlocal){

    hipfftHandle plan;

    int nFFTs = nlocal / Ng;

    #ifdef doublePrecision

    if (hipfftPlan1d(&plan, Ng, HIPFFT_Z2Z, nFFTs) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;	
    }

    if (hipfftExecZ2Z(plan, (cudafftPrecision*)data[0], (cudafftPrecision*)data[0], HIPFFT_FORWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecZ2Z Forward failed\n");
        return;	
    }

    #else

    if (hipfftPlan1d(&plan, Ng, HIPFFT_C2C, nFFTs) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;	
    }

    if (hipfftExecC2C(plan, (cudafftPrecision*)data[0], (cudafftPrecision*)data[0], HIPFFT_FORWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecC2C Forward failed\n");
        return;	
    }
    #endif

    hipDeviceSynchronize();

}

extern void inverse_1d_fft(fftPrecision** data, int Ng, int nlocal){

    hipfftHandle plan;

    int nFFTs = nlocal / Ng;

    #ifdef doublePrecision

    if (hipfftPlan1d(&plan, Ng, HIPFFT_Z2Z, nFFTs) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;	
    }

    if (hipfftExecZ2Z(plan, (cudafftPrecision*)data[0], (cudafftPrecision*)data[0], HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecZ2Z Forward failed\n");
        return;	
    }

    #else

    if (hipfftPlan1d(&plan, Ng, HIPFFT_C2C, nFFTs) != HIPFFT_SUCCESS){
        printf("CUFFT error: Plan creation failed\n");
        return;	
    }

    if (hipfftExecC2C(plan, (cudafftPrecision*)data[0], (cudafftPrecision*)data[0], HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
        printf("CUFFT error: ExecC2C Forward failed\n");
        return;	
    }

    #endif

    hipDeviceSynchronize();

    }

__global__
void scale_fft(fftPrecision* data, int Ng, int nlocal){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    fftPrecision scale = (fftPrecision)(Ng * Ng * Ng);

    if (idx < nlocal){

        data[idx*2] = data[idx*2] / scale;
        data[idx*2 + 1] = data[idx*2 + 1] / scale;

    }

}

extern void launch_scale_fft(fftPrecision** data, int Ng, int nlocal, int blockSize){

    int numBlocks = (nlocal + blockSize - 1) / blockSize;

    scale_fft<<<numBlocks,blockSize>>>(data[0],Ng,nlocal);

    hipDeviceSynchronize();

}