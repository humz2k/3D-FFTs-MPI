#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "kernels.h"

extern void copy_h2d(fftPrecision** dest, fftPrecision* source, int nlocal){
    if (hipMemcpy(dest[0],source,nlocal * 2 * sizeof(fftPrecision),hipMemcpyHostToDevice) != hipSuccess){
        printf("Memcpy Error h2d >> %s\n",hipGetErrorString(hipGetLastError()));
    }
}

extern void copy_d2h(fftPrecision* dest, fftPrecision** source, int nlocal){
    if (hipMemcpy(dest,source[0],nlocal * 2 * sizeof(fftPrecision),hipMemcpyDeviceToHost) != hipSuccess){
        printf("Memcpy Error d2h >> %s\n",hipGetErrorString(hipGetLastError()));
    }
}