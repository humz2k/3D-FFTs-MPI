#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_utils.h"

extern void initialize_cuda(fftPrecision** d_myGridCellsBuff1, fftPrecision** d_myGridCellsBuff2, int nlocal){
    hipFree(0);
    if (hipMalloc(&(d_myGridCellsBuff1[0]),nlocal * 2 * sizeof(fftPrecision)) != hipSuccess){
        printf("MALLOC ERROR >> %s\n", hipGetErrorString(hipGetLastError()));
    };
    if (hipMalloc(&(d_myGridCellsBuff2[0]),nlocal * 2 * sizeof(fftPrecision)) != hipSuccess){
        printf("MALLOC ERROR >> %s\n", hipGetErrorString(hipGetLastError()));
    };
}

extern void finalize_cuda(fftPrecision** d_myGridCellsBuff1, fftPrecision** d_myGridCellsBuff2){
    if (hipFree(d_myGridCellsBuff1[0]) != hipSuccess){
        printf("hipFree Error >> %s\n",hipGetErrorString(hipGetLastError()));
    }
    if (hipFree(d_myGridCellsBuff2[0]) != hipSuccess){
        printf("hipFree Error >> %s\n",hipGetErrorString(hipGetLastError()));
    }
}

extern void util_copy_h2d(fftPrecision** dest, fftPrecision* source, int nlocal){
    if (hipMemcpy(dest[0],source,nlocal * 2 * sizeof(fftPrecision),hipMemcpyHostToDevice) != hipSuccess){
        printf("Memcpy Error h2d >> %s\n",hipGetErrorString(hipGetLastError()));
    }
}

extern void util_copy_d2h(fftPrecision* dest, fftPrecision** source, int nlocal){
    if (hipMemcpy(dest,source[0],nlocal * 2 * sizeof(fftPrecision),hipMemcpyDeviceToHost) != hipSuccess){
        printf("Memcpy Error d2h >> %s\n",hipGetErrorString(hipGetLastError()));
    }
}