#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_utils.h"

extern void initialize_cuda(fftPrecision** d_myGridCellsBuff1, fftPrecision** d_myGridCellsBuff2, int nlocal){
    hipFree(0);
    if (hipMalloc(&(d_myGridCellsBuff1[0]),nlocal * 2 * sizeof(fftPrecision)) != hipSuccess){
        printf("MALLOC ERROR >> %s\n", hipGetErrorString(hipGetLastError()));
    };
    if (hipMalloc(&(d_myGridCellsBuff2[0]),nlocal * 2 * sizeof(fftPrecision)) != hipSuccess){
        printf("MALLOC ERROR >> %s\n", hipGetErrorString(hipGetLastError()));
    };
}

extern void finalize_cuda(fftPrecision** d_myGridCellsBuff1, fftPrecision** d_myGridCellsBuff2){
    if (hipFree(d_myGridCellsBuff1[0]) != hipSuccess){
        printf("hipFree Error >> %s\n",hipGetErrorString(hipGetLastError()));
    }
    if (hipFree(d_myGridCellsBuff2[0]) != hipSuccess){
        printf("hipFree Error >> %s\n",hipGetErrorString(hipGetLastError()));
    }
}