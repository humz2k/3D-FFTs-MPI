#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


extern "C" {
    void initialize_cuda(float** d_myGridCellsBuff1, float** d_myGridCellsBuff2, int nlocal, float* myGridCellsBuff1){
        hipFree(0);
        if (hipMalloc(&(d_myGridCellsBuff1[0]),nlocal * 2 * sizeof(float)) != hipSuccess){
            printf("MALLOC ERROR >> %s\n", hipGetErrorString(hipGetLastError()));
        };
        if (hipMalloc(&(d_myGridCellsBuff2[0]),nlocal * 2 * sizeof(float)) != hipSuccess){
            printf("MALLOC ERROR >> %s\n", hipGetErrorString(hipGetLastError()));
        };
    }
}

extern "C" {
    void copy_h2d(float** dest, float* source, int nlocal){
        if (hipMemcpy(dest[0],source,nlocal * 2 * sizeof(float),hipMemcpyHostToDevice) != hipSuccess){
            printf("Memcpy Error h2d >> %s\n",hipGetErrorString(hipGetLastError()));
        }

    }
}

extern "C" {
    void copy_d2h(float* dest, float** source, int nlocal){
        if (hipMemcpy(dest,source[0],nlocal * 2 * sizeof(float),hipMemcpyDeviceToHost) != hipSuccess){
            printf("Memcpy Error d2h >> %s\n",hipGetErrorString(hipGetLastError()));
        }
    }
}

extern "C" {
    void finalize_cuda(float** d_myGridCellsBuff1, float** d_myGridCellsBuff2){
        if (hipFree(d_myGridCellsBuff1[0]) != hipSuccess){
            printf("hipFree Error >> %s\n",hipGetErrorString(hipGetLastError()));
        }
        if (hipFree(d_myGridCellsBuff2[0]) != hipSuccess){
            printf("hipFree Error >> %s\n",hipGetErrorString(hipGetLastError()));
        }
    }
}