#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


extern "C" {
    void initialize_cuda(float* d_myGridCellsBuff1, float* d_myGridCellsBuff2, int nlocal){
        hipFree(0);
        hipMalloc(&d_myGridCellsBuff1,nlocal * 2 * sizeof(float));
        hipMalloc(&d_myGridCellsBuff2,nlocal * 2 * sizeof(float));
    }
}

extern "C" {
    void copy_h2d(float* source, float* dest, int nlocal){
        hipMemcpy(dest,source,nlocal * 2 * sizeof(float),hipMemcpyHostToDevice);

        float* test = (float*) malloc(nlocal * sizeof(float) * 2);

        hipMemcpy(test,dest,nlocal*sizeof(float)*2,hipMemcpyDeviceToHost);

        printf("TEST %f\n",test[2]);

        free(test);

    }
}

extern "C" {
    void copy_d2h(float* source, float* dest, int nlocal){
        hipMemcpy(dest,source,nlocal * 2 * sizeof(float),hipMemcpyDeviceToHost);
    }
}

extern "C" {
    void finalize_cuda(float* d_myGridCellsBuff1, float* d_myGridCellsBuff2){
        hipFree(d_myGridCellsBuff1);
        hipFree(d_myGridCellsBuff2);
    }
}