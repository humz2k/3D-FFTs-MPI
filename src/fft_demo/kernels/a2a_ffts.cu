
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>

extern "C" {

    void forward_1d_fft(float** data, int Ng, int nlocal){

        hipfftHandle plan;

        int nFFTs = nlocal / Ng;

        if (hipfftPlan1d(&plan, Ng, HIPFFT_C2C, nFFTs) != HIPFFT_SUCCESS){
            printf("CUFFT error: Plan creation failed\n");
            return;	
        }

        if (hipfftExecC2C(plan, (hipfftComplex*)data[0], (hipfftComplex*)data[0], HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            printf("CUFFT error: ExecC2C Forward failed\n");
            return;	
        }

        hipDeviceSynchronize();

        /*if (cufftExecC2C(plan, (cufftComplex*)data[0], (cufftComplex*)data[0], CUFFT_INVERSE) != CUFFT_SUCCESS){
            printf("CUFFT error: ExecC2C Forward failed\n");
            return;	
        }*/

    }

}

extern "C" {

    void inverse_1d_fft(float** data, int Ng, int nlocal){

        hipfftHandle plan;

        int nFFTs = nlocal / Ng;

        if (hipfftPlan1d(&plan, Ng, HIPFFT_C2C, nFFTs) != HIPFFT_SUCCESS){
            printf("CUFFT error: Plan creation failed\n");
            return;	
        }

        if (hipfftExecC2C(plan, (hipfftComplex*)data[0], (hipfftComplex*)data[0], HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
            printf("CUFFT error: ExecC2C Forward failed\n");
            return;	
        }

        hipDeviceSynchronize();

        /*if (cufftExecC2C(plan, (cufftComplex*)data[0], (cufftComplex*)data[0], CUFFT_INVERSE) != CUFFT_SUCCESS){
            printf("CUFFT error: ExecC2C Forward failed\n");
            return;	
        }*/

    }

}

__global__
void scale_fft(float* data, int Ng, int nlocal){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    float scale = (float)(Ng * Ng * Ng);

    if (idx < nlocal){

        data[idx*2] = data[idx*2] / scale;
        data[idx*2 + 1] = data[idx*2 + 1] / scale;

    }

}

extern "C" {

    void launch_scale_fft(float** data, int Ng, int nlocal, int blockSize){

        int numBlocks = (nlocal + blockSize - 1) / blockSize;

        scale_fft<<<numBlocks,blockSize>>>(data[0],Ng,nlocal);

        hipDeviceSynchronize();

    }

}