
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>

extern "C" {

    void forward_1d_fft(float** data, int Ng, int nlocal){

        hipfftHandle plan;

        int nFFTs = nlocal / Ng;

        if (hipfftPlan1d(&plan, Ng, HIPFFT_C2C, nFFTs) != HIPFFT_SUCCESS){
            printf("CUFFT error: Plan creation failed\n");
            return;	
        }

        if (hipfftExecC2C(plan, (hipfftComplex*)data[0], (hipfftComplex*)data[0], HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            printf("CUFFT error: ExecC2C Forward failed\n");
            return;	
        }

        /*if (cufftExecC2C(plan, (cufftComplex*)data[0], (cufftComplex*)data[0], CUFFT_INVERSE) != CUFFT_SUCCESS){
            printf("CUFFT error: ExecC2C Forward failed\n");
            return;	
        }*/

    }

}